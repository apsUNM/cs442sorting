#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <time.h>
#include <math.h>

#include "mpi_cannon.hpp"


void gpu_merge(float* A, float* B, float** grid){
  //############################ PASSED PARAMETERS #############################
    // float *A = h_A;
    // float *B = h_B;
    // float **grid = h_grid;


  //######################### INITIALIZING VARIABLES #############################

    //CPU
    //Findout how many processes
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    //GPU
    //Set Size of Blocks and Threads
    dim3 dimBlock(32,32);
    int grid_dim = ceil(n /32.0);
    dim3 dimGrid(grid_dim,grid_dim);
}

//copy_to_cpu_cannon(h_A, h_B, h_C_copy, n, sq_num_procs, rank_row, rank_col);
void copy_to_cpu_cannon(float* A, float* B, float* C,
        int n, int sq_num_procs, int rank_row, int rank_col)
{
  //############################ PASSED PARAMETERS ###############################
    // float *A = h_A;
    // float *B = h_B;
    // float *C = h_C_copy;
    // int n = n;
    // int sq_num_procs = sq_num_procs
    // int rank_row = rank_row
    // int rank_col = rank_col



  //######################### INITIALIZING VARIABLES #############################

    //CPU
    //Findout how many processes
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    //GPU
    //Set Size of Blocks and Threads
    dim3 dimBlock(32,32);
    int grid_dim = ceil(n /32.0);
    dim3 dimGrid(grid_dim,grid_dim);

    //CPU
    //Identify size of array
    int size = n*n;
    int fsize = size * sizeof(float);

    //CPU
    //Create an array of total size
    float* send_A = new float[size];
    float* recv_A = new float[size];
    float* send_B = new float[size];
    float* recv_B = new float[size];


    //GPU
    //Setup Arrays
    float* d_send_A=nullptr;
    float* d_recv_A=nullptr;
    float* d_send_B=nullptr;
    float* d_recv_B=nullptr;

    float* d_C=nullptr;
    
    //GPU
    //hipMalloc(void** buffer, int size)
    hipMalloc((void**)&d_send_A,fsize);
    hipMalloc((void**)&d_recv_A,fsize);
    hipMalloc((void**)&d_send_B,fsize);
    hipMalloc((void**)&d_recv_B,fsize);
    hipMalloc((void**)&d_C,fsize);



    //CPU
    //Idk picking processors to shift with
    int send_proc_A, send_proc_B;
    int recv_proc_A, recv_proc_B;
    int tag_a = 1234;
    int tag_b = 4321;

    //CPU
    //Sets Array c to 0
    memset(C, 0, size*sizeof(float));
    hipMemcpy(d_C,C,fsize,hipMemcpyHostToDevice);

  //########################## CANNON INITIAL SHIFT ############################## 
    
    //CPU
    //Sets the send/recv procs to correct values for initial shift
    get_init_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    //Sends the array A to next proc
    communicate(send_proc_A, recv_proc_A, tag_a, size, 
            rank_row && rank_col / rank_row % 2 == 0, A, recv_A);
    //Sends the array B to next proc
    communicate(send_proc_B, recv_proc_B, tag_b, size, 
            rank_col && rank_row / rank_col % 2 == 0, B, recv_B);
    //Does Matrix Multiplication of A and B and puts it in C
    //matmat(n, recv_A, recv_B, C);
    
    
    //GPU
    //CopyPortions to GPU
    hipMemcpy(d_recv_A,recv_A,fsize,hipMemcpyHostToDevice);
    hipMemcpy(d_recv_B,recv_B,fsize,hipMemcpyHostToDevice);
    //Do Multiplication on GPU
    matrixMultKernel<<<dimGrid,dimBlock>>>(n,d_recv_A,d_recv_B,d_C);



  //########################## CANNON GENERAL SHIFT ##############################

    //CPU
    // Send and recv A and B from neighborhing processes in proc grid
    get_rotation_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    for (int i = 1; i < sq_num_procs; i++)
    {
        //Pointer Swap between send_A and recv_A, Swap between send_B and recv_B
        swap(&send_A, &recv_A, &send_B, &recv_B);
        
        communicate(send_proc_A, recv_proc_A, tag_a, size, rank_col % 2 == 0,
                send_A, recv_A);
        communicate(send_proc_B, recv_proc_B, tag_b, size, rank_row % 2 == 0,
                send_B, recv_B);
        //matmat(n, recv_A, recv_B, C);
        hipMemcpy(d_recv_A,recv_A,fsize,hipMemcpyHostToDevice);
        hipMemcpy(d_recv_B,recv_B,fsize,hipMemcpyHostToDevice);
        matrixMultKernel<<<dimGrid,dimBlock>>>(n,d_recv_A,d_recv_B,d_C);
    }

  //################################ CLEANUP #####################################

    //GPU
    //Return C
    hipMemcpy(C,d_C,fsize,hipMemcpyDeviceToHost);

    //CPU
    //Frees Allocated Arrays
    delete[] send_A;
    delete[] recv_A;
    delete[] send_B;
    delete[] recv_B;

    //GPU
    //Free Allocated Arrays
    //hipFree(void** buffer)
    hipFree(d_send_A);
    hipFree(d_recv_A);
    hipFree(d_send_B);
    hipFree(d_recv_B);
    hipFree(d_C);
}



//cuda_aware_cannon(h_A, h_B, h_C_cuda, n, sq_num_procs, rank_row, rank_col);
void cuda_aware_cannon(float* A, float* B, float* C,
        int n, int sq_num_procs, int rank_row, int rank_col)
{
  //############################ PASSED PARAMETERS ###############################
    // float *A = h_A;
    // float *B = h_B;
    // float *C = h_C_copy;
    // int n = n;
    // int sq_num_procs = sq_num_procs
    // int rank_row = rank_row
    // int rank_col = rank_col



  //######################### INITIALIZING VARIABLES #############################

    //CPU
    //Findout how many processes
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    //GPU
    //Set Size of Blocks and Threads
    dim3 dimBlock(32,32);
    int grid_dim = ceil(n /32.0);
    dim3 dimGrid(grid_dim,grid_dim);

    //CPU
    //Identify size of array
    int size = n*n;
    int fsize = size * sizeof(float);

    //CPU
    //Create an array of total size
    float* send_A = new float[size];
    float* recv_A = new float[size];
    float* send_B = new float[size];
    float* recv_B = new float[size];


    //GPU
    //Setup Arrays
    float* d_send_A=nullptr;
    float* d_recv_A=nullptr;
    float* d_send_B=nullptr;
    float* d_recv_B=nullptr;

    float* d_C=nullptr;
    
    //GPU
    //hipMalloc(void** buffer, int size)
    hipMalloc((void**)&d_send_A,fsize);
    hipMalloc((void**)&d_recv_A,fsize);
    hipMalloc((void**)&d_send_B,fsize);
    hipMalloc((void**)&d_recv_B,fsize);
    hipMalloc((void**)&d_C,fsize);



    //CPU
    //Idk picking processors to shift with
    int send_proc_A, send_proc_B;
    int recv_proc_A, recv_proc_B;
    int tag_a = 1234;
    int tag_b = 4321;

    //CPU
    //Sets Array c to 0
    memset(C, 0, size*sizeof(float));
    hipMemcpy(d_send_A,A,fsize,hipMemcpyHostToDevice);
    hipMemcpy(d_send_B,B,fsize,hipMemcpyHostToDevice);
    hipMemcpy(d_C,C,fsize,hipMemcpyHostToDevice);

  //########################## CANNON INITIAL SHIFT ############################## 
    
    //CPU
    //Sets the send/recv procs to correct values for initial shift
    get_init_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    //Sends the array A to next proc
    cuda_aware_comm(send_proc_A, recv_proc_A, tag_a, size, 
            rank_row && rank_col / rank_row % 2 == 0,hipMemcpyDeviceToDevice, d_send_A, d_recv_A);
    //Sends the array B to next proc
    cuda_aware_comm(send_proc_B, recv_proc_B, tag_b, size, 
            rank_col && rank_row / rank_col % 2 == 0,hipMemcpyDeviceToDevice, d_send_B, d_recv_B);
    //Does Matrix Multiplication of A and B and puts it in C
    //matmat(n, recv_A, recv_B, C);
    
    
    //GPU
    //CopyPortions to GPU
    //hipMemcpy(d_recv_A,recv_A,fsize,hipMemcpyHostToDevice);
    //hipMemcpy(d_recv_B,recv_B,fsize,hipMemcpyHostToDevice);
    //Do Multiplication on GPU
    matrixMultKernel<<<dimGrid,dimBlock>>>(n,d_recv_A,d_recv_B,d_C);



  //########################## CANNON GENERAL SHIFT ##############################

    //CPU
    // Send and recv A and B from neighborhing processes in proc grid
    get_rotation_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    for (int i = 1; i < sq_num_procs; i++)
    {
        //Pointer Swap between send_A and recv_A, Swap between send_B and recv_B
        swap(&d_send_A, &d_recv_A, &d_send_B, &d_recv_B);
        
        cuda_aware_comm(send_proc_A, recv_proc_A, tag_a, size, rank_col % 2 == 0,
                hipMemcpyDeviceToDevice,d_send_A, d_recv_A);
        cuda_aware_comm(send_proc_B, recv_proc_B, tag_b, size, rank_row % 2 == 0,
                hipMemcpyDeviceToDevice,d_send_B, d_recv_B);
        //matmat(n, recv_A, recv_B, C);
        //hipMemcpy(d_recv_A,recv_A,fsize,hipMemcpyHostToDevice);
        //hipMemcpy(d_recv_B,recv_B,fsize,hipMemcpyHostToDevice);
        matrixMultKernel<<<dimGrid,dimBlock>>>(n,d_recv_A,d_recv_B,d_C);
    }

  //################################ CLEANUP #####################################

    //GPU
    //Return C
    hipMemcpy(C,d_C,fsize,hipMemcpyDeviceToHost);

    //CPU
    //Frees Allocated Arrays
    delete[] send_A;
    delete[] recv_A;
    delete[] send_B;
    delete[] recv_B;

    //GPU
    //Free Allocated Arrays
    //hipFree(void** buffer)
    hipFree(d_send_A);
    hipFree(d_recv_A);
    hipFree(d_send_B);
    hipFree(d_recv_B);
    hipFree(d_C);
}





